#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2020 Neka-Nat
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 **/
#include "cupoch/knn/kdtree_flann.h"
#include "cupoch/geometry/pointcloud.h"
#include "cupoch/geometry/geometry_utils.h"
#include "cupoch/utility/console.h"

#include <thrust/logical.h>

using namespace cupoch;
using namespace cupoch::geometry;

namespace {

struct compute_vertex_degree_functor {
    compute_vertex_degree_functor(int *indices, int min_points, int max_edges)
        : indices_(indices), min_points_(min_points), max_edges_(max_edges){};
    int *indices_;
    const int min_points_;
    const int max_edges_;
    __device__ int operator()(size_t idx) const {
        int count = 0;
        for (int k = 0; k < max_edges_; k++) {
            if (indices_[idx * max_edges_ + k] >= 0) {
                if (indices_[idx * max_edges_ + k] == idx) {
                    indices_[idx * max_edges_ + k] = -1;
                } else {
                    count++;
                }
            }
        }
        if (count >= min_points_) return count;
        for (int k = 0; k < max_edges_; k++) {
            indices_[idx * max_edges_ + k] = -1;
        }
        return 0;
    }
};

struct bfs_functor {
    bfs_functor(const int *vertex_degrees,
                const int *exscan_vd,
                const int *indices,
                int *xa,
                bool *fa)
        : vertex_degrees_(vertex_degrees),
          exscan_vd_(exscan_vd),
          indices_(indices),
          xa_(xa),
          fa_(fa){};
    const int *vertex_degrees_;
    const int *exscan_vd_;
    const int *indices_;
    int *xa_;
    bool *fa_;
    __device__ void operator()(size_t idx) const {
        if (fa_[idx] == 1) {
            fa_[idx] = false;
            xa_[idx] = 1;
            const int vd = __ldg(&vertex_degrees_[idx]);
            for (int i = 0; i < vd; i++) {
                const int ev = __ldg(&exscan_vd_[idx]);
                const int index = __ldg(&indices_[ev + i]);
                if (xa_[index] == 0) {
                    fa_[index] = true;
                }
            }
        }
    }
};

struct set_label_functor {
    set_label_functor(const int *xa, int cluster, int *clusters, int *visited)
        : xa_(xa), cluster_(cluster), clusters_(clusters), visited_(visited){};
    const int *xa_;
    const int cluster_;
    int *clusters_;
    int *visited_;
    __device__ void operator()(size_t idx) const {
        if (xa_[idx] == 1) {
            clusters_[idx] = cluster_;
            visited_[idx] = 1;
        }
    }
};

}  // namespace

// https://www.sciencedirect.com/science/article/pii/S1877050913003438
std::unique_ptr<utility::device_vector<int>> PointCloud::ClusterDBSCAN(float eps,
                                                      size_t min_points,
                                                      bool print_progress,
                                                      size_t max_edges) const {
    // precompute all neighbours
    utility::LogDebug("Precompute Neighbours");
    utility::ConsoleProgressBar progress_bar(
            points_.size(), "Precompute Neighbours", print_progress);

    const size_t n_pt = points_.size();
    // Graph construction
    utility::device_vector<int> vertex_degrees(n_pt);
    utility::device_vector<int> exscan_vd(n_pt);
    utility::device_vector<int> indices;
    utility::device_vector<float> distances;
    knn::KDTreeFlann kdtree(ConvertVector3fVectorRef(*this));
    kdtree.SearchRadius(points_, eps, max_edges + 1, indices, distances);
    compute_vertex_degree_functor vd_func(
            thrust::raw_pointer_cast(indices.data()), min_points,
            max_edges + 1);
    thrust::transform(thrust::make_counting_iterator<size_t>(0),
                      thrust::make_counting_iterator(n_pt),
                      vertex_degrees.begin(), vd_func);
    thrust::exclusive_scan(utility::exec_policy(0),
                           vertex_degrees.begin(), vertex_degrees.end(),
                           exscan_vd.begin(), 0);
    auto end = thrust::remove_if(utility::exec_policy(0),
                                 indices.begin(), indices.end(),
                                 [] __device__(int idx) { return idx < 0; });
    indices.resize(thrust::distance(indices.begin(), end));

    // Cluster identification
    int cluster = 0;
    utility::device_vector<int> visited(n_pt, 0);
    utility::pinned_host_vector<int> h_visited(n_pt, 0);
    auto clusters = std::make_unique<utility::device_vector<int>>(n_pt, -1);
    utility::device_vector<int> xa(n_pt);
    utility::device_vector<bool> fa(n_pt);
    for (int i = 0; i < n_pt; i++) {
        ++progress_bar;
        if (h_visited[i] != 1) {
            thrust::fill_n(make_tuple_iterator(visited.begin() + i,
                                               clusters->begin() + i),
                           1, thrust::make_tuple(1, cluster));
            thrust::fill(make_tuple_begin(xa, fa), make_tuple_end(xa, fa),
                         thrust::make_tuple(0, 0));
            fa[i] = true;
            while (!thrust::any_of(fa.begin(), fa.end(), thrust::identity<bool>())) {
                bfs_functor bfs_func(
                        thrust::raw_pointer_cast(vertex_degrees.data()),
                        thrust::raw_pointer_cast(exscan_vd.data()),
                        thrust::raw_pointer_cast(indices.data()),
                        thrust::raw_pointer_cast(xa.data()),
                        thrust::raw_pointer_cast(fa.data()));
                thrust::for_each(thrust::make_counting_iterator<size_t>(0),
                                 thrust::make_counting_iterator(n_pt),
                                 bfs_func);
            }
            set_label_functor sl_func(thrust::raw_pointer_cast(xa.data()),
                                      cluster,
                                      thrust::raw_pointer_cast(clusters->data()),
                                      thrust::raw_pointer_cast(visited.data()));
            thrust::for_each(thrust::make_counting_iterator<size_t>(0),
                             thrust::make_counting_iterator(n_pt), sl_func);
            copy_device_to_host(visited, h_visited);
            cudaSafeCall(hipDeviceSynchronize());
            cluster++;
        }
    }
    return clusters;
}